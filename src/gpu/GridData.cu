#include "gpu/GridData.h"
#include <stdexcept>
#include <hip/hip_runtime.h>

void allocateGridDataGPU(GridData& gd, size_t N, size_t numCells)
{
    // Prima libera eventuali buffer gi� allocati
    if (gd.particleGridIndices) hipFree(gd.particleGridIndices);
    if (gd.particleArrayIndices) hipFree(gd.particleArrayIndices);
    if (gd.cellStartIndices) hipFree(gd.cellStartIndices);
    if (gd.cellEndIndices) hipFree(gd.cellEndIndices);

    // Aggiorna dimensioni
    gd.numCells = numCells;

    // Allocazione dei buffer per i boid
    if (hipMalloc(&gd.particleGridIndices, N * sizeof(int)) != hipSuccess)
        throw std::runtime_error("hipMalloc failed: particleGridIndices");
    if (hipMalloc(&gd.particleArrayIndices, N * sizeof(int)) != hipSuccess)
        throw std::runtime_error("hipMalloc failed: particleArrayIndices");

    // Allocazione dei buffer per le celle
    if (hipMalloc(&gd.cellStartIndices, numCells * sizeof(int)) != hipSuccess)
        throw std::runtime_error("hipMalloc failed: cellStartIndices");
    if (hipMalloc(&gd.cellEndIndices, numCells * sizeof(int)) != hipSuccess)
        throw std::runtime_error("hipMalloc failed: cellEndIndices");

    // Inizializza le celle a -1 (vuote)
    hipMemset(gd.cellStartIndices, -1, numCells * sizeof(int));
    hipMemset(gd.cellEndIndices, -1, numCells * sizeof(int));
}

void freeGridDataGPU(GridData& gd)
{
    if (gd.particleGridIndices) {
        hipFree(gd.particleGridIndices);
        gd.particleGridIndices = nullptr;
    }
    if (gd.particleArrayIndices) {
        hipFree(gd.particleArrayIndices);
        gd.particleArrayIndices = nullptr;
    }
    if (gd.cellStartIndices) {
        hipFree(gd.cellStartIndices);
        gd.cellStartIndices = nullptr;
    }
    if (gd.cellEndIndices) {
        hipFree(gd.cellEndIndices);
        gd.cellEndIndices = nullptr;
    }

    gd.numCells = 0;
}
