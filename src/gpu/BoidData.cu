#include "../gpu/BoidData.h"
#include <vector>
#include <hip/hip_runtime.h>
#include <iostream>

// --- Helper per controllare errori CUDA ---
#define CUDA_CHECK(err) \
    if(err != hipSuccess) { \
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl; \
        exit(EXIT_FAILURE); \
    }

// --- Allocazione GPU ---
void allocateBoidDataGPU(BoidData& bd, size_t N) {
    bd.N = N;
    CUDA_CHECK(hipMalloc(&bd.posX, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&bd.posY, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&bd.velX, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&bd.velY, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&bd.driftX, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&bd.driftY, N * sizeof(float)));

    CUDA_CHECK(hipMalloc(&bd.scale, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&bd.influence, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&bd.type, N * sizeof(int)));
    CUDA_CHECK(hipMalloc(&bd.age, N * sizeof(int)));
    CUDA_CHECK(hipMalloc(&bd.birthTime, N * sizeof(float)));

    CUDA_CHECK(hipMalloc(&bd.colorR, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&bd.colorG, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&bd.colorB, N * sizeof(float)));

    for (int i = 0; i < 5; ++i)
        CUDA_CHECK(hipMalloc(&bd.debugX[i], N * sizeof(float)));
    for (int i = 0; i < 5; ++i)
        CUDA_CHECK(hipMalloc(&bd.debugY[i], N * sizeof(float)));
}

// --- Copia CPU -> GPU ---
void copyBoidsToGPU(const std::vector<Boid>& cpuBoids, BoidData& bd) {
    size_t N = cpuBoids.size();

    std::vector<float> posX(N), posY(N), velX(N), velY(N), driftX(N), driftY(N);
    std::vector<float> scale(N), influence(N), birthTime(N);
    std::vector<int> type(N), age(N);
    std::vector<float> colorR(N), colorG(N), colorB(N);
    std::vector<float> debugX[5], debugY[5];
    for (int i = 0; i < 5; i++) { debugX[i].resize(N); debugY[i].resize(N); }

    for (size_t i = 0; i < N; i++) {
        const Boid& b = cpuBoids[i];
        posX[i] = b.position.x;
        posY[i] = b.position.y;
        velX[i] = b.velocity.x;
        velY[i] = b.velocity.y;
        driftX[i] = b.drift.x;
        driftY[i] = b.drift.y;

        scale[i] = b.scale;
        influence[i] = b.influence;
        type[i] = static_cast<int>(b.type);
        age[i] = b.age;
        birthTime[i] = b.birthTime;

        colorR[i] = b.color.r;
        colorG[i] = b.color.g;
        colorB[i] = b.color.b;

        for (int j = 0; j < 5; j++) {
            debugX[j][i] = b.debugVectors[j].x;
            debugY[j][i] = b.debugVectors[j].y;
        }
    }

    CUDA_CHECK(hipMemcpy(bd.posX, posX.data(), N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(bd.posY, posY.data(), N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(bd.velX, velX.data(), N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(bd.velY, velY.data(), N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(bd.driftX, driftX.data(), N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(bd.driftY, driftY.data(), N * sizeof(float), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMemcpy(bd.scale, scale.data(), N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(bd.influence, influence.data(), N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(bd.type, type.data(), N * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(bd.age, age.data(), N * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(bd.birthTime, birthTime.data(), N * sizeof(float), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMemcpy(bd.colorR, colorR.data(), N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(bd.colorG, colorG.data(), N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(bd.colorB, colorB.data(), N * sizeof(float), hipMemcpyHostToDevice));

    for (int j = 0; j < 5; j++) {
        CUDA_CHECK(hipMemcpy(bd.debugX[j], debugX[j].data(), N * sizeof(float), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(bd.debugY[j], debugY[j].data(), N * sizeof(float), hipMemcpyHostToDevice));
    }
}

// --- Copia GPU -> CPU ---
void copyBoidsToCPU(BoidData& bd, std::vector<Boid>& cpuBoids) {
    size_t N = bd.N;
    cpuBoids.resize(N);

    std::vector<float> posX(N), posY(N), velX(N), velY(N), driftX(N), driftY(N);
    std::vector<float> scale(N), influence(N), birthTime(N);
    std::vector<int> type(N), age(N);
    std::vector<float> colorR(N), colorG(N), colorB(N);
    std::vector<float> debugX[5], debugY[5];
    for (int i = 0; i < 5; i++) { debugX[i].resize(N); debugY[i].resize(N); }

    CUDA_CHECK(hipMemcpy(posX.data(), bd.posX, N * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(posY.data(), bd.posY, N * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(velX.data(), bd.velX, N * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(velY.data(), bd.velY, N * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(driftX.data(), bd.driftX, N * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(driftY.data(), bd.driftY, N * sizeof(float), hipMemcpyDeviceToHost));

    CUDA_CHECK(hipMemcpy(scale.data(), bd.scale, N * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(influence.data(), bd.influence, N * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(type.data(), bd.type, N * sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(age.data(), bd.age, N * sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(birthTime.data(), bd.birthTime, N * sizeof(float), hipMemcpyDeviceToHost));

    CUDA_CHECK(hipMemcpy(colorR.data(), bd.colorR, N * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(colorG.data(), bd.colorG, N * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(colorB.data(), bd.colorB, N * sizeof(float), hipMemcpyDeviceToHost));

    for (int j = 0; j < 5; j++) {
        CUDA_CHECK(hipMemcpy(debugX[j].data(), bd.debugX[j], N * sizeof(float), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(debugY[j].data(), bd.debugY[j], N * sizeof(float), hipMemcpyDeviceToHost));
    }

    for (size_t i = 0; i < N; i++) {
        cpuBoids[i].position = { posX[i], posY[i] };
        cpuBoids[i].velocity = { velX[i], velY[i] };
        cpuBoids[i].drift = { driftX[i], driftY[i] };
        cpuBoids[i].scale = scale[i];
        cpuBoids[i].influence = influence[i];
        cpuBoids[i].type = static_cast<BoidType>(type[i]);
        cpuBoids[i].age = age[i];
        cpuBoids[i].birthTime = birthTime[i];
        cpuBoids[i].color = { colorR[i], colorG[i], colorB[i] };
        for (int j = 0; j < 5; j++)
            cpuBoids[i].debugVectors[j] = { debugX[j][i], debugY[j][i] };
    }
}

// --- Libera GPU ---
void freeBoidDataGPU(BoidData& bd) {
    hipFree(bd.posX); hipFree(bd.posY);
    hipFree(bd.velX); hipFree(bd.velY);
    hipFree(bd.driftX); hipFree(bd.driftY);

    hipFree(bd.scale); hipFree(bd.influence);
    hipFree(bd.type); hipFree(bd.age); hipFree(bd.birthTime);

    hipFree(bd.colorR); hipFree(bd.colorG); hipFree(bd.colorB);

    for (int i = 0; i < 5; i++) { hipFree(bd.debugX[i]); hipFree(bd.debugY[i]); }
}