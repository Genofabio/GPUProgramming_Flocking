#include "../gpu/BoidData.h"
#include <vector>
#include <hip/hip_runtime.h>
#include <iostream>

// --- Helper per controllare errori CUDA ---
#define CUDA_CHECK(err) \
    if(err != hipSuccess) { \
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl; \
        exit(EXIT_FAILURE); \
    }

// --- Allocazione GPU ---
void allocateBoidDataGPU(BoidData& bd, size_t N) {
    bd.N = N;

    CUDA_CHECK(hipMalloc(&bd.posX, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&bd.posY, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&bd.velX, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&bd.velY, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&bd.driftX, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&bd.driftY, N * sizeof(float)));

    CUDA_CHECK(hipMalloc(&bd.scale, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&bd.influence, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&bd.type, N * sizeof(int)));
    CUDA_CHECK(hipMalloc(&bd.age, N * sizeof(int)));
    CUDA_CHECK(hipMalloc(&bd.birthTime, N * sizeof(float)));

    CUDA_CHECK(hipMalloc(&bd.velChangeX, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&bd.velChangeY, N * sizeof(float)));

    CUDA_CHECK(hipMalloc(&bd.colorR, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&bd.colorG, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&bd.colorB, N * sizeof(float)));

    // --- Nuovo buffer per rotazioni ---
    CUDA_CHECK(hipMalloc(&bd.rotations, N * sizeof(float)));

    for (int i = 0; i < 5; ++i) {
        CUDA_CHECK(hipMalloc(&bd.debugX[i], N * sizeof(float)));
        CUDA_CHECK(hipMalloc(&bd.debugY[i], N * sizeof(float)));
    }

    // --- Buffer _sorted (senza drift_sorted) ---
    CUDA_CHECK(hipMalloc(&bd.posX_sorted, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&bd.posY_sorted, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&bd.velX_sorted, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&bd.velY_sorted, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&bd.velChangeX_sorted, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&bd.velChangeY_sorted, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&bd.scale_sorted, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&bd.influence_sorted, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&bd.type_sorted, N * sizeof(int)));
    CUDA_CHECK(hipMalloc(&bd.colorR_sorted, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&bd.colorG_sorted, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&bd.colorB_sorted, N * sizeof(float)));
}

// --- Copia CPU -> GPU ---
void copyBoidsToGPU(const std::vector<Boid>& cpuBoids, BoidData& bd) {
    size_t N = cpuBoids.size();

    std::vector<float> posX(N), posY(N), velX(N), velY(N);
    std::vector<float> driftX(N), driftY(N);
    std::vector<float> scale(N), influence(N), birthTime(N);
    std::vector<int> type(N), age(N);
    std::vector<float> colorR(N), colorG(N), colorB(N);
    std::vector<float> debugX[5], debugY[5];
    for (int i = 0; i < 5; i++) { debugX[i].resize(N); debugY[i].resize(N); }

    for (size_t i = 0; i < N; i++) {
        const Boid& b = cpuBoids[i];
        posX[i] = b.position.x; posY[i] = b.position.y;
        velX[i] = b.velocity.x; velY[i] = b.velocity.y;
        driftX[i] = b.drift.x; driftY[i] = b.drift.y;

        scale[i] = b.scale;
        influence[i] = b.influence;
        type[i] = static_cast<int>(b.type);
        age[i] = b.age;
        birthTime[i] = b.birthTime;

        colorR[i] = b.color.r; colorG[i] = b.color.g; colorB[i] = b.color.b;

        for (int j = 0; j < 5; j++) {
            debugX[j][i] = b.debugVectors[j].x;
            debugY[j][i] = b.debugVectors[j].y;
        }
    }

    CUDA_CHECK(hipMemcpy(bd.posX, posX.data(), N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(bd.posY, posY.data(), N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(bd.velX, velX.data(), N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(bd.velY, velY.data(), N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(bd.driftX, driftX.data(), N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(bd.driftY, driftY.data(), N * sizeof(float), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMemcpy(bd.scale, scale.data(), N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(bd.influence, influence.data(), N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(bd.type, type.data(), N * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(bd.age, age.data(), N * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(bd.birthTime, birthTime.data(), N * sizeof(float), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMemcpy(bd.colorR, colorR.data(), N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(bd.colorG, colorG.data(), N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(bd.colorB, colorB.data(), N * sizeof(float), hipMemcpyHostToDevice));

    for (int j = 0; j < 5; j++) {
        CUDA_CHECK(hipMemcpy(bd.debugX[j], debugX[j].data(), N * sizeof(float), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(bd.debugY[j], debugY[j].data(), N * sizeof(float), hipMemcpyHostToDevice));
    }

    // Copia iniziale nei buffer _sorted
    CUDA_CHECK(hipMemcpy(bd.posX_sorted, posX.data(), N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(bd.posY_sorted, posY.data(), N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(bd.velX_sorted, velX.data(), N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(bd.velY_sorted, velY.data(), N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(bd.velChangeX_sorted, bd.velChangeX, N * sizeof(float), hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipMemcpy(bd.velChangeY_sorted, bd.velChangeY, N * sizeof(float), hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipMemcpy(bd.scale_sorted, scale.data(), N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(bd.influence_sorted, influence.data(), N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(bd.type_sorted, type.data(), N * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(bd.colorR_sorted, colorR.data(), N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(bd.colorG_sorted, colorG.data(), N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(bd.colorB_sorted, colorB.data(), N * sizeof(float), hipMemcpyHostToDevice));
}

// --- Libera GPU ---
void freeBoidDataGPU(BoidData& bd) {
    hipFree(bd.posX); hipFree(bd.posY);
    hipFree(bd.velX); hipFree(bd.velY);
    hipFree(bd.driftX); hipFree(bd.driftY);

    hipFree(bd.scale); hipFree(bd.influence);
    hipFree(bd.type); hipFree(bd.age); hipFree(bd.birthTime);

    hipFree(bd.colorR); hipFree(bd.colorG); hipFree(bd.colorB);

    hipFree(bd.velChangeX); hipFree(bd.velChangeY);

    hipFree(bd.rotations);

    for (int i = 0; i < 5; i++) { hipFree(bd.debugX[i]); hipFree(bd.debugY[i]); }

    hipFree(bd.posX_sorted); hipFree(bd.posY_sorted);
    hipFree(bd.velX_sorted); hipFree(bd.velY_sorted);
    hipFree(bd.velChangeX_sorted); hipFree(bd.velChangeY_sorted);
    hipFree(bd.scale_sorted); hipFree(bd.influence_sorted);
    hipFree(bd.type_sorted);
    hipFree(bd.colorR_sorted); hipFree(bd.colorG_sorted); hipFree(bd.colorB_sorted);
}
