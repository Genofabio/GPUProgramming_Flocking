#include "hip/hip_runtime.h"
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <gpu/CudaKernels.cuh>
#include <algorithm>

// --- Dichiarazioni in memoria costante ---
__constant__ int d_width;
__constant__ int d_height;

// Parametri boid generali
__constant__ float d_maxSpeed;
__constant__ float d_slowDownFactor;

// Distanze per le regole
__constant__ float d_cohesionDistance;
__constant__ float d_separationDistance;
__constant__ float d_alignmentDistance;
__constant__ float d_borderDistance;
__constant__ float d_predatorFearDistance;
__constant__ float d_predatorChaseDistance;
__constant__ float d_predatorSeparationDistance;
__constant__ float d_predatorEatDistance;
__constant__ float d_leaderInfluenceDistance;
__constant__ float d_desiredLeaderDistance;
__constant__ float d_wallRepulsionDistance;

// Pesi per le regole
__constant__ float d_cohesionScale;
__constant__ float d_separationScale;
__constant__ float d_alignmentScale;
__constant__ float d_borderScale;
__constant__ float d_predatorFearScale;
__constant__ float d_predatorChaseScale;
__constant__ float d_predatorSeparationScale;
__constant__ float d_borderAlertDistance;
__constant__ float d_leaderInfluenceScale;
__constant__ float d_wallRepulsionScale;

// Parametri specifici
__constant__ float d_mateDistance;
__constant__ int   d_mateThreshold;
__constant__ int   d_matingAge;
__constant__ float d_predatorBoostRadius;
__constant__ float d_allyRadius;

void setSimulationParamsOnGPU(int width, int height, const BoidParams& params) {
    hipMemcpyToSymbol(HIP_SYMBOL(d_width), &width, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_height), &height, sizeof(int));

    // Generali
    hipMemcpyToSymbol(HIP_SYMBOL(d_maxSpeed), &params.maxSpeed, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_slowDownFactor), &params.slowDownFactor, sizeof(float));

    // Distanze
    hipMemcpyToSymbol(HIP_SYMBOL(d_cohesionDistance), &params.cohesionDistance, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_separationDistance), &params.separationDistance, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_alignmentDistance), &params.alignmentDistance, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_borderDistance), &params.borderDistance, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_predatorFearDistance), &params.predatorFearDistance, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_predatorChaseDistance), &params.predatorChaseDistance, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_predatorSeparationDistance), &params.predatorSeparationDistance, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_predatorEatDistance), &params.predatorEatDistance, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_leaderInfluenceDistance), &params.leaderInfluenceDistance, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_desiredLeaderDistance), &params.desiredLeaderDistance, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_wallRepulsionDistance), &params.wallRepulsionDistance, sizeof(float));

    // Pesi
    hipMemcpyToSymbol(HIP_SYMBOL(d_cohesionScale), &params.cohesionScale, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_separationScale), &params.separationScale, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_alignmentScale), &params.alignmentScale, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_borderScale), &params.borderScale, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_predatorFearScale), &params.predatorFearScale, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_predatorChaseScale), &params.predatorChaseScale, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_predatorSeparationScale), &params.predatorSeparationScale, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_borderAlertDistance), &params.borderAlertDistance, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_leaderInfluenceScale), &params.leaderInfluenceScale, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_wallRepulsionScale), &params.wallRepulsionScale, sizeof(float));

    // Specifici
    hipMemcpyToSymbol(HIP_SYMBOL(d_mateDistance), &params.mateDistance, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_mateThreshold), &params.mateThreshold, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_matingAge), &params.matingAge, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_predatorBoostRadius), &params.predatorBoostRadius, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_allyRadius), &params.allyRadius, sizeof(float));
}

__global__ void computeForcesKernelAggressive(
    int N,
    const float* posX_sorted, const float* posY_sorted,
    const float* velX_sorted, const float* velY_sorted,
    const float* influence_sorted,
    const int* gridCellStartIndices,
    const int* gridCellEndIndices,
    int gridResolutionX, int gridResolutionY,
    float cellWidth,
    float* outVelChangeX, float* outVelChangeY,
    int numWalls,
    const float2* wallPositions,
    const int* type_sorted)  // x,y start/end concatenati
{
    extern __shared__ float shMem[];
    float* shPosX = shMem;
    float* shPosY = shPosX + blockDim.x;
    float* shVelX = shPosY + blockDim.x;
    float* shVelY = shVelX + blockDim.x;
    float* shInfluence = shVelY + blockDim.x;

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    float px = posX_sorted[i];
    float py = posY_sorted[i];

    float cohX = 0.f, cohY = 0.f;
    float sepX = 0.f, sepY = 0.f;
    float aliX = 0.f, aliY = 0.f;
    int neighborCount = 0;
    float totalWeight = 0.f;

    int col = (int)(px / cellWidth);
    col = (col < 0) ? 0 : ((col >= gridResolutionX) ? gridResolutionX - 1 : col);

    int row = (int)(py / cellWidth);
    row = (row < 0) ? 0 : ((row >= gridResolutionY) ? gridResolutionY - 1 : row);

    int dr[4] = {
        0,
        ((px - col * cellWidth) > 0.5f * cellWidth) ? 1 : -1,
        0,
        ((px - col * cellWidth) > 0.5f * cellWidth) ? 1 : -1
    };
    int dc[4] = {
        0,
        0,
        ((py - row * cellWidth) > 0.5f * cellWidth) ? 1 : -1,
        ((py - row * cellWidth) > 0.5f * cellWidth) ? 1 : -1
    };

    for (int q = 0; q < 4; ++q) {
        if (type_sorted[i] == 2) break;

        int neighRow = row + dr[q];
        int neighCol = col + dc[q];
        if (neighRow < 0 || neighRow >= gridResolutionY) continue;
        if (neighCol < 0 || neighCol >= gridResolutionX) continue;

        int cellIdx = neighCol + neighRow * gridResolutionX;
        int startIdx = gridCellStartIndices[cellIdx];
        int endIdx = gridCellEndIndices[cellIdx];
        if (startIdx == -1) continue;

        int tileSize = endIdx - startIdx + 1;

        // --- tiling in shared memory ---
        for (int offset = 0; offset < tileSize; offset += blockDim.x) {
            int tid = threadIdx.x + offset;
            if (tid < tileSize) {
                int idxTile = startIdx + tid;
                shPosX[threadIdx.x] = posX_sorted[idxTile];
                shPosY[threadIdx.x] = posY_sorted[idxTile];
                shVelX[threadIdx.x] = velX_sorted[idxTile];
                shVelY[threadIdx.x] = velY_sorted[idxTile];
                shInfluence[threadIdx.x] = influence_sorted[idxTile];
            }
            __syncthreads();

            int limit = (tileSize - offset < blockDim.x) ? (tileSize - offset) : blockDim.x;
            for (int j = 0; j < limit; ++j) {
                int globalIdx = startIdx + offset + j;
                if (globalIdx == i) continue;

                float dx = shPosX[j] - px;
                float dy = shPosY[j] - py;
                float dist = sqrtf(dx * dx + dy * dy);

                if (dist < d_cohesionDistance) {
                    cohX += shPosX[j];
                    cohY += shPosY[j];
                    neighborCount++;
                }
                if (dist < d_separationDistance && dist > 0.f) {
                    sepX += (px - shPosX[j]) / dist;
                    sepY += (py - shPosY[j]) / dist;
                }
                if (dist < d_alignmentDistance && type_sorted[globalIdx] != 2) {
                    aliX += shVelX[j] * shInfluence[j];
                    aliY += shVelY[j] * shInfluence[j];
                    totalWeight += shInfluence[j];
                }
            }
            __syncthreads();
        }
    }

    if (neighborCount > 0) {
        cohX = (cohX / neighborCount - px) * d_cohesionScale;
        cohY = (cohY / neighborCount - py) * d_cohesionScale;
    }
    if (totalWeight > 0.f) {
        aliX = (aliX / totalWeight) * d_alignmentScale;
        aliY = (aliY / totalWeight) * d_alignmentScale;
    }

    sepX *= d_separationScale;
    sepY *= d_separationScale;

    // --- Border forces ---
    float borderX = 0.f, borderY = 0.f;
    if (px < d_borderAlertDistance) borderX += (d_borderAlertDistance - px);
    if ((d_width - px) < d_borderAlertDistance) borderX -= (d_borderAlertDistance - (d_width - px));
    if (py < d_borderAlertDistance) borderY += (d_borderAlertDistance - py);
    if ((d_height - py) < d_borderAlertDistance) borderY -= (d_borderAlertDistance - (d_height - py));
    borderX *= 0.2f;
    borderY *= 0.2f;

    // --- Repulsione dai muri ---
    float wallRepX = 0.f, wallRepY = 0.f;

    float lookAhead = 30.0f;

    // Calcola direzione normalizzata della velocit�
    float velLen = sqrtf(velX_sorted[i] * velX_sorted[i] + velY_sorted[i] * velY_sorted[i]);
    float dirX = (velLen > 0.0001f) ? velX_sorted[i] / velLen : 0.f;
    float dirY = (velLen > 0.0001f) ? velY_sorted[i] / velLen : 0.f;

    for (int w = 0; w < numWalls; ++w) {
        float2 start = wallPositions[2 * w];     // punto inizio muro
        float2 end = wallPositions[2 * w + 1]; // punto fine muro

        // distanza punto-muro: proiezione del boid sul segmento
        float dx = px - start.x;
        float dy = py - start.y;
        float wallLenX = end.x - start.x;
        float wallLenY = end.y - start.y;
        float wallLenSq = wallLenX * wallLenX + wallLenY * wallLenY;

        float t = fmaxf(0.f, fminf(1.f, (dx * wallLenX + dy * wallLenY) / wallLenSq));

        float closestX = start.x + t * wallLenX;
        float closestY = start.y + t * wallLenY;

        float distX = px - closestX;
        float distY = py - closestY;
        float dist = sqrtf(distX * distX + distY * distY);

        if (dist < d_wallRepulsionDistance && dist > 0.001f) {
            // safe lookahead (per evitare instabilit� troppo vicino al muro)
            float safeLookAhead = fmaxf(0.001f, fminf(lookAhead, dist - 0.2f));
            float probeX = px + dirX * safeLookAhead;
            float probeY = py + dirY * safeLookAhead;

            // direzione "via dal muro"
            float awayX = probeX - closestX;
            float awayY = probeY - closestY;
            float awayLen = sqrtf(awayX * awayX + awayY * awayY);
            if (awayLen > 0.0001f) {
                awayX /= awayLen;
                awayY /= awayLen;
            }

            // forza con fattore quadratico e divisione per distanza
            float factor = (d_wallRepulsionDistance - dist) / dist;
            float force = factor * factor * d_wallRepulsionScale;

            wallRepX += awayX * force;
            wallRepY += awayY * force;
        }
    }

    outVelChangeX[i] = cohX + sepX + aliX + borderX + wallRepX;
    outVelChangeY[i] = cohY + sepY + aliY + borderY + wallRepY;
}

__global__ void kernComputeIndices(
    int N,
    float* posX, float* posY,
    int* particleGridIndices,
    int* particleArrayIndices,
    int gridResolutionX, int gridResolutionY,
    float gridMinX, float gridMinY,
    float cellWidth)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    int cellX = (int)floorf((posX[i] - gridMinX) / cellWidth);
    int cellY = (int)floorf((posY[i] - gridMinY) / cellWidth);

    cellX = (cellX < 0) ? 0 : ((cellX >= gridResolutionX) ? gridResolutionX - 1 : cellX);
    cellY = (cellY < 0) ? 0 : ((cellY >= gridResolutionY) ? gridResolutionY - 1 : cellY);

    int cellIndex = cellX + cellY * gridResolutionX;

    particleGridIndices[i] = cellIndex;
    particleArrayIndices[i] = i;
}


__global__ void kernIdentifyCellStartEnd(
    int N,
    int* particleGridIndices,
    int* gridCellStartIndices,
    int* gridCellEndIndices)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    int currentCell = particleGridIndices[i];
    int prevCell = (i > 0) ? particleGridIndices[i - 1] : -1;
    int nextCell = (i < N - 1) ? particleGridIndices[i + 1] : -1;

    if (currentCell != prevCell) {
        gridCellStartIndices[currentCell] = i;
    }
    if (currentCell != nextCell) {
        gridCellEndIndices[currentCell] = i;
    }
}


__global__ void kernApplyVelocityChangeSorted(
    int N,
    const float* velChangeX_sorted, const float* velChangeY_sorted,
    float* posX, float* posY,
    float* velX, float* velY,
    const int* particleArrayIndices,
    const int* type_sorted,  // aggiungi questo array
    float dt)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    int origIdx = particleArrayIndices[i];

    // Applica cambiamento di velocit�
    velX[origIdx] += velChangeX_sorted[i] * d_slowDownFactor;
    velY[origIdx] += velChangeY_sorted[i] * d_slowDownFactor;

    // Calcola velocit�
    float speed = sqrtf(velX[origIdx] * velX[origIdx] + velY[origIdx] * velY[origIdx]);

    // Limita la velocit� massima
    if (speed > d_maxSpeed) {
        velX[origIdx] = (velX[origIdx] / speed) * d_maxSpeed;
        velY[origIdx] = (velY[origIdx] / speed) * d_maxSpeed;
        speed = d_maxSpeed;  // aggiorna la velocit�
    }

    // **Mantieni velocit� minima per i leader**
    if (type_sorted[i] == 2) {  // 2 = leader
        float minSpeed = d_maxSpeed / 1.1f; // definisci questa costante
        if (speed < minSpeed) {
            velX[origIdx] = (velX[origIdx] / speed) * minSpeed;
            velY[origIdx] = (velY[origIdx] / speed) * minSpeed;
        }
    }

    // Aggiorna posizione
    posX[origIdx] += velX[origIdx] * dt;
    posY[origIdx] += velY[origIdx] * dt;
}



__global__ void kernComputeRotations(
    int N, const float* velX, const float* velY, float* rotations)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    float angle = atan2f(velY[i], velX[i]);   // radianti
    angle = angle * (180.0f / 3.14159265f);   // in gradi
    rotations[i] = angle + 270.0f;            // offset per orientamento modello
}

__global__ void kernIntegratePositions(
    int N, float dt,
    float* posX, float* posY,
    const float* velX, const float* velY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    posX[i] += velX[i] * dt;
    posY[i] += velY[i] * dt;
}

__global__ void kernReorderData(
    int N,
    const float* posX, const float* posY,
    const float* velX, const float* velY,
    const float* scale, const float* influence,
    const int* type,
    const float* colorR, const float* colorG, const float* colorB,
    const float* velChangeX, const float* velChangeY,
    const int* particleArrayIndices,
    float* posX_sorted, float* posY_sorted,
    float* velX_sorted, float* velY_sorted,
    float* scale_sorted, float* influence_sorted,
    int* type_sorted,
    float* colorR_sorted, float* colorG_sorted, float* colorB_sorted,
    float* velChangeX_sorted, float* velChangeY_sorted)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    int srcIdx = particleArrayIndices[i];

    posX_sorted[i] = posX[srcIdx];
    posY_sorted[i] = posY[srcIdx];
    velX_sorted[i] = velX[srcIdx];
    velY_sorted[i] = velY[srcIdx];
    scale_sorted[i] = scale[srcIdx];
    influence_sorted[i] = influence[srcIdx];
    type_sorted[i] = type[srcIdx];
    colorR_sorted[i] = colorR[srcIdx];
    colorG_sorted[i] = colorG[srcIdx];
    colorB_sorted[i] = colorB[srcIdx];
    velChangeX_sorted[i] = velChangeX[srcIdx];
    velChangeY_sorted[i] = velChangeY[srcIdx];
}

__global__ void copyRenderDataKernel(
    int N,
    const float* posX, const float* posY,
    const float* rotations,
    const float* colorR, const float* colorG, const float* colorB,
    const float* scale,
    glm::vec2* outPositions,
    float* outRotations,
    glm::vec3* outColors,
    float* outScales)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    outPositions[i] = { posX[i], posY[i] };
    outRotations[i] = rotations[i];
    outColors[i] = { colorR[i], colorG[i], colorB[i] };
    outScales[i] = scale[i];
}

__global__ void computeLeaderFollowKernel(
    int N,
    const float* posX_sorted,
    const float* posY_sorted,
    const float* velX_sorted,
    const float* velY_sorted,
    const int* type_sorted,
    float* velChangeX_sorted,
    float* velChangeY_sorted)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    float px = posX_sorted[i];
    float py = posY_sorted[i];

    float deltaX = 0.f;
    float deltaY = 0.f;

    if (type_sorted[i] == 2) {
        // Leader: evita altri leader
        for (int j = 0; j < N; ++j) {
            if (j == i) continue;
            if (type_sorted[j] != 2) continue;

            float dx = px - posX_sorted[j];
            float dy = py - posY_sorted[j];
            float dist = sqrtf(dx * dx + dy * dy);
            if (dist < d_desiredLeaderDistance && dist > 0.001f) {
                float factor = (d_desiredLeaderDistance - dist) / dist;
                deltaX += dx * factor * 0.8f;
                deltaY += dy * factor * 0.8f;
            }
        }
    }
    else {
        // Follower: allineamento + coesione verso leader pi� vicino
        float closestDist = 1e20f;
        int closestIdx = -1;

        for (int j = 0; j < N; ++j) {
            if (type_sorted[j] != 2) continue; // solo leader
            float dx = posX_sorted[j] - px;
            float dy = posY_sorted[j] - py;
            float dist = sqrtf(dx * dx + dy * dy);
            if (dist < d_leaderInfluenceDistance && dist < closestDist) {
                closestDist = dist;
                closestIdx = j;
            }
        }

        if (closestIdx >= 0) {
            // Coesione verso leader
            float dx = posX_sorted[closestIdx] - px;
            float dy = posY_sorted[closestIdx] - py;
            float norm = (d_leaderInfluenceDistance - closestDist) / d_leaderInfluenceDistance;
            float cohesionWeight = norm * norm;
            deltaX += dx * cohesionWeight * d_leaderInfluenceScale * 0.5f;
            deltaY += dy * cohesionWeight * d_leaderInfluenceScale * 0.5f;

            // Allineamento con velocit� del leader
            float alignWeight = 0.5f; // puoi regolare
            deltaX += (velX_sorted[closestIdx] - 0.0f) * alignWeight;
            deltaY += (velY_sorted[closestIdx] - 0.0f) * alignWeight;
        }
    }

    velChangeX_sorted[i] += deltaX;
    velChangeY_sorted[i] += deltaY;
}
