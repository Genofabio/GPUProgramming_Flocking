#include "hip/hip_runtime.h"
#include <set>
#include <map>
#include <iostream>
#include <cmath>
#include <algorithm>
#include <random>

#include <gpu/Simulation.cuh>
#include <utility/ResourceManager.h>
#include <gpu/BoidData.h>
#include <core/Boid.h>
#include <gpu/cuda_kernel.cuh>
#include <hip/hip_runtime.h>

Simulation::Simulation(unsigned int width, unsigned int height)
    : keys()
    , width(width)
    , height(height)
    , wallGrid(10, 15, static_cast<float>(width), static_cast<float>(height), 1)
    , boidRenderer(nullptr)
    , wallRenderer(nullptr)
    , gridRenderer(nullptr)
    , textRenderer(nullptr)
    , rng(std::random_device{}())
    , dist(-1.0f, 1.0f)
{
    // Inizializzazione dei parametri di boid
    params.maxSpeed = 100.0f;          // come prima, velocit� naturale
    params.slowDownFactor = 0.3f;      // frenata normale

    // Distanze (adattate a maxBoidDistance = 150)
    params.cohesionDistance = 60.0f;   // leggermente pi� piccolo per stare entro griglia
    params.separationDistance = 20.0f; // mantenere sicurezza nelle collisioni
    params.alignmentDistance = 40.0f;  // simile all�originale ma compatto
    params.borderDistance = 80.0f;     // non serve cos� grande
    params.predatorFearDistance = 100.0f;
    params.predatorChaseDistance = 120.0f;
    params.predatorSeparationDistance = 50.0f;
    params.predatorEatDistance = 5.0f; // come prima

    // Scale (forza delle regole)
    params.cohesionScale = 0.1f;      // originale
    params.separationScale = 2.2f;     // come prima
    params.alignmentScale = 0.19f;     // originale
    params.borderScale = 0.3f;         // coerente
    params.predatorFearScale = 0.8f;   // originale
    params.predatorChaseScale = 0.12f;
    params.predatorSeparationScale = 2.0f;
    params.borderAlertDistance = height / 5.0f;

    // Social/extra
    params.leaderInfluenceDistance = 120.0f; // ridotto proporzionalmente
    params.leaderInfluenceScale = 1.0f;
    params.mateDistance = 10.0f;
    params.mateThreshold = 200;
    params.matingAge = 6;
    params.predatorBoostRadius = 70.0f;
    params.desiredLeaderDistance = 150.0f;
    params.allyRadius = 50.0f;

    // Griglia boid basata sulla distanza massima di interazione
    float maxBoidDistance = 300.0f;//std::max({ params.cohesionDistance, params.separationDistance, params.alignmentDistance, params.borderDistance, params.predatorFearDistance, params.predatorChaseDistance, params.predatorSeparationDistance, params.predatorEatDistance });
    int nCols = static_cast<int>(std::ceil(width / maxBoidDistance));
    int nRows = static_cast<int>(std::ceil(height / maxBoidDistance));
    boidGrid = UniformBoidGrid(nRows, nCols, static_cast<float>(width), static_cast<float>(height));
}

Simulation::~Simulation()
{
    delete wallRenderer;
    delete boidRenderer;
    delete textRenderer;
    delete gridRenderer;
    delete vectorRenderer;
}

void Simulation::init()
{
    // Shader setup
    ResourceManager::LoadShader("shaders/boid_shader.vert", "shaders/boid_shader.frag", nullptr, "boid");
    ResourceManager::LoadShader("shaders/wall_shader.vert", "shaders/wall_shader.frag", nullptr, "wall");
    ResourceManager::LoadShader("shaders/grid_shader.vert", "shaders/grid_shader.frag", nullptr, "grid");
    ResourceManager::LoadShader("shaders/text_shader.vert", "shaders/text_shader.frag", nullptr, "text");
    ResourceManager::LoadShader("shaders/vector_shader.vert", "shaders/vector_shader.frag", nullptr, "vector");

    glm::mat4 projection = glm::ortho(0.0f, static_cast<float>(width), 0.0f, static_cast<float>(height), -1.0f, 1.0f);

    ResourceManager::GetShader("boid").Use().SetMatrix4("projection", projection);
    ResourceManager::GetShader("wall").Use().SetMatrix4("projection", projection);
    ResourceManager::GetShader("grid").Use().SetMatrix4("projection", projection);
    ResourceManager::GetShader("text").Use().SetInteger("text", 0);
    ResourceManager::GetShader("text").SetMatrix4("projection", projection);
    ResourceManager::GetShader("vector").Use().SetMatrix4("projection", projection);

    // Renderer setup
    boidRenderer = new BoidRenderer(ResourceManager::GetShader("boid"));
    wallRenderer = new WallRenderer(ResourceManager::GetShader("wall"));
    gridRenderer = new GridRenderer(ResourceManager::GetShader("grid"));
    textRenderer = new TextRenderer(ResourceManager::GetShader("text"));
    textRenderer->loadFont("resources/fonts/Roboto/Roboto-Regular.ttf", 24);
    vectorRenderer = new VectorRenderer(ResourceManager::GetShader("vector"));

    // Initialize boids
    initLeaders(0);
    initPrey(200);
    initPredators(0);

    // Allocate and copy boid data to GPU
    if (!boidDataInitialized) {
        allocateBoidDataGPU(gpuBoids, boids.size());
        copyBoidsToGPU(boids, gpuBoids);
        boidDataInitialized = true;
    }

    // Initialize walls
    initWalls(0);
}

void Simulation::update(float dt)
{
    profiler.start();
    currentTime += dt;

    std::vector<glm::vec2> positions;
    for (const auto& b : boids)
        positions.push_back(b.position);

    boidGrid.updateCells(positions);

    size_t N = boids.size();
    std::vector<glm::vec2> velocityChanges(N, glm::vec2(0.0f));

    // 1. Calcola tutte le forze
    computeForces(velocityChanges);

    // 2. Applica le velocit� ai boid
    applyVelocity(dt, velocityChanges);

    // 3. Controlla quali prede sono state mangiate e le rimuove
    checkEatenPrey();

    // 4. Gestisce l'accoppiamento e lo spawn di nuovi boid
    spawnNewBoids();

    profiler.log("update", profiler.stop());
}

void Simulation::render()
{
    profiler.start();

    // 1. Draw grid
    glLineWidth(1.0f);
    gridRenderer->draw(wallGrid, glm::vec3(0.2f, 0.2f, 0.2f));
    //gridRenderer->draw(boidGrid, glm::vec3(0.6f, 0.6f, 0.6f));

    // 2. Draw boids
    for (const Boid& b : boids) {
        float angle = glm::degrees(atan2(b.velocity.y, b.velocity.x)) + 270.0f;
        boidRenderer->draw(b.position, angle, b.color, 10.0f * b.scale);
    }

    // 3. Draw walls
    glLineWidth(3.0f);
    for (const Wall& w : walls)
        wallRenderer->draw(w, glm::vec3(0.25f, 0.88f, 0.82f));

    profiler.log("render", profiler.stop());

    // 4. Draw debug vectors
    //for (Boid& b : boids) {
    //    if (b.type == LEADER || b.type == PREDATOR) {
    //        glm::vec2 start = b.position;

    //        vectorRenderer->DrawVector(start, start + b.debugVectors[0] * 20.0f, glm::vec3(1.0f, 1.0f, 0.0f)); // giallo
    //        vectorRenderer->DrawVector(start, start + b.debugVectors[1] * 20.0f, glm::vec3(1.0f, 0.0f, 0.0f)); // rosso
    //        vectorRenderer->DrawVector(start, start + b.debugVectors[2] * 20.0f, glm::vec3(0.05f, 0.8f, 0.7f)); // ciano
    //        vectorRenderer->DrawVector(start, start + b.debugVectors[3] * 20.0f, glm::vec3(0.0f, 0.0f, 1.0f)); // blu
    //        //vectorRenderer->DrawVector(start, start + b.debugVectors[4] * 20.0f, glm::vec3(1.0f, 0.0f, 1.0f)); // magenta
    //    }
    //}

    // 5. Draw HUD / stats
    float margin = 10.0f;
    float scale = 0.7f;
    glm::vec3 color(0.9f, 0.9f, 0.3f);

    double fps = profiler.getCurrentFPS();
    if (fps > 0.0)
        textRenderer->draw("FPS: " + std::to_string(static_cast<int>(fps)), margin, height - margin - 20.0f, scale, color);

    textRenderer->draw("Boids: " + std::to_string(boids.size()), margin, height - margin - 40.0f, scale, color);
}

void Simulation::processInput(float dt) {}

void Simulation::updateStats(float dt) {
    profiler.updateFrameStats(dt);
}

void Simulation::saveProfilerCSV(const std::string& path) {
    profiler.saveCSV(path);
}


// === HELPER Init ===
void Simulation::initLeaders(int count)
{
    for (int i = 0; i < count; ++i) {
        Boid b;
        b.position = glm::vec2(static_cast<float>(rand() % width), static_cast<float>(rand() % height));
        b.velocity = glm::vec2(static_cast<float>(rand() % 201 - 100) * 0.5f, static_cast<float>(rand() % 201 - 100) * 0.5f);
        b.type = BoidType::LEADER;
        b.age = 10;
        b.scale = 1.7f;
        b.color = glm::vec3(0.9f, 0.9f, 0.2f);
        b.drift = glm::vec2(0);
        boids.push_back(b);
    }
}

void Simulation::initPrey(int count)
{
    std::uniform_int_distribution<int> ageDist(0, 6);
    std::uniform_real_distribution<float> offsetDist(0.0f, 30.0f);

    for (int i = 0; i < count; ++i) {
        Boid b;
        b.position = glm::vec2(static_cast<float>(rand() % width), static_cast<float>(rand() % height));
        b.velocity = glm::vec2(static_cast<float>(rand() % 201 - 100) * 0.5f, static_cast<float>(rand() % 201 - 100) * 0.5f);
        b.type = BoidType::PREY;
        b.birthTime = currentTime + offsetDist(rng);
        b.age = ageDist(rng);
        float t = b.age / 10.0f;
        b.scale = 1.0f + 0.04f * b.age;
        b.color = glm::mix(glm::vec3(0.2f, 0.2f, 0.9f), glm::vec3(0.05f, 0.8f, 0.7f), t);
        b.influence = 0.8f + 0.04f * b.age;
        boids.push_back(b);
    }
}

void Simulation::initPredators(int count)
{
    for (int i = 0; i < count; ++i) {
        Boid b;
        b.position = glm::vec2(static_cast<float>(rand() % width), static_cast<float>(rand() % height));
        b.velocity = glm::vec2(static_cast<float>(rand() % 201 - 100) * 0.5f, static_cast<float>(rand() % 201 - 100) * 0.5f);
        b.type = BoidType::PREDATOR;
        b.age = 10;
        b.scale = 1.9f;
        b.color = glm::vec3(0.9f, 0.2f, 0.2f);
        b.drift = glm::vec2(0);
        boids.push_back(b);
    }
}

void Simulation::initWalls(int count)
{
    auto candidates = wallGrid.cellEdges;
    std::shuffle(candidates.begin(), candidates.end(), rng);

    std::set<std::pair<std::pair<int, int>, std::pair<int, int>>> usedEdges;
    std::map<std::pair<int, int>, std::pair<bool, bool>> vertexOccupancy;

    int added = 0;
    for (const auto& edge : candidates) {
        if (added >= count) break;

        auto p1 = std::make_pair(int(edge.first.x), int(edge.first.y));
        auto p2 = std::make_pair(int(edge.second.x), int(edge.second.y));
        if (p2 < p1) std::swap(p1, p2);

        bool horizontal = (p1.second == p2.second);
        if (usedEdges.count({ p1, p2 })) continue;
        if ((horizontal && (vertexOccupancy[p1].second || vertexOccupancy[p2].second)) ||
            (!horizontal && (vertexOccupancy[p1].first || vertexOccupancy[p2].first))) continue;

        Wall w({ edge.first, edge.second }, height / 11.0f, 3.0f);
        walls.push_back(w);  // popola direttamente il membro della classe

        usedEdges.insert({ p1, p2 });
        if (horizontal) {
            vertexOccupancy[p1].first = true;
            vertexOccupancy[p2].first = true;
        }
        else {
            vertexOccupancy[p1].second = true;
            vertexOccupancy[p2].second = true;
        }

        added++;
    }
}

// === HELPER Update ===
void Simulation::computeForces(std::vector<glm::vec2>& velocityChanges) {
    int N = static_cast<int>(boids.size());

    int threads = 256;
    int blocks = (N + threads - 1) / threads;

    computeForcesKernel << <blocks, threads >> > (
        N,
        gpuBoids.posX, gpuBoids.posY,
        gpuBoids.velX, gpuBoids.velY,
        gpuBoids.influence,
        gpuBoids.type,
        gpuBoids.velX, // qui puoi scrivere direttamente su velX o su buffer separato
        gpuBoids.velY,
        params.cohesionDistance, params.cohesionScale,
        params.separationDistance, params.separationScale,
        params.alignmentDistance, params.alignmentScale
        );

    hipDeviceSynchronize();
}

//void Simulation::computeForces(std::vector<glm::vec2>& velocityChanges)
//{
//    size_t N = boids.size();
//
//    for (size_t i = 0; i < N; ++i) {
//        Boid& b = boids[i];
//        glm::vec2 totalChange(0.0f);
//
//        BoidRules::computeBoidUpgrade(b, currentTime);
//
//        int col = static_cast<int>(b.position.x / wallGrid.cellWidth);
//        int row = static_cast<int>(b.position.y / wallGrid.cellHeight);
//
//        // Ottieni boid vicini
//        std::vector<size_t> nearbyIndices = boidGrid.getNearbyBoids(b.position);
//
//        // Suddividi i vicini per tipo
//        std::vector<size_t> nearbyPrey, nearbyPredators, nearbyLeaders, nearbyAllies;
//        for (size_t idx : nearbyIndices) {
//            const Boid& other = boids[idx];
//            if (&b == &other) continue;
//
//            switch (other.type) {
//            case PREY:   nearbyPrey.push_back(idx); nearbyAllies.push_back(idx); break;
//            case PREDATOR: nearbyPredators.push_back(idx); break;
//            case LEADER: nearbyLeaders.push_back(idx); break;
//            }
//        }
//
//        switch (b.type) {
//        case PREY:
//            totalChange += BoidRules::computeCohesion(b, boids, nearbyPrey, params.cohesionDistance, params.cohesionScale);
//            totalChange += BoidRules::computeSeparation(b, boids, nearbyPrey, params.separationDistance, params.separationScale);
//            totalChange += BoidRules::computeAlignment(b, boids, nearbyPrey, params.alignmentDistance, params.alignmentScale);
//            totalChange += BoidRules::computeFollowLeaders(b, boids, nearbyLeaders, params.leaderInfluenceDistance, params.leaderInfluenceScale);
//            totalChange += BoidRules::computeBorderRepulsion(b.position, static_cast<float>(width), static_cast<float>(height), params.borderAlertDistance);
//            totalChange += BoidRules::computeWallRepulsion(b.position, b.velocity, walls);
//            totalChange += BoidRules::computeEvadePredators(b, boids, nearbyPredators, nearbyAllies, params.predatorFearDistance, params.predatorFearScale, params.allyRadius);
//            break;
//
//        case PREDATOR:
//            b.debugVectors[0] = BoidRules::computeChasePrey(i, boids, nearbyPrey, params.predatorChaseDistance, params.predatorChaseScale, params.predatorBoostRadius);
//            b.debugVectors[1] = BoidRules::computePredatorSeparation(b, boids, nearbyPredators, params.predatorSeparationDistance) * params.predatorSeparationScale;
//            b.debugVectors[2] = BoidRules::computeBorderRepulsion(b.position, static_cast<float>(width), static_cast<float>(height), params.borderAlertDistance);
//            b.debugVectors[3] = BoidRules::computeWallRepulsion(b.position, b.velocity, walls);
//
//            totalChange = b.debugVectors[0] + b.debugVectors[1] + b.debugVectors[2] + b.debugVectors[3];
//            break;
//
//        case LEADER:
//            b.debugVectors[0] = BoidRules::computeLeaderSeparation(b, boids, nearbyLeaders, params.desiredLeaderDistance);
//            b.debugVectors[1] = BoidRules::computeBorderRepulsion(b.position, static_cast<float>(width), static_cast<float>(height), params.borderAlertDistance);
//            b.debugVectors[2] = BoidRules::computeWallRepulsion(b.position, b.velocity, walls);
//            b.debugVectors[3] = BoidRules::computeEvadePredators(b, boids, nearbyPredators, nearbyAllies, params.predatorFearDistance, params.predatorFearScale, params.allyRadius);
//
//            totalChange = b.debugVectors[0] + b.debugVectors[1] + b.debugVectors[2] + b.debugVectors[3];
//            break;
//        }
//
//        velocityChanges[i] = totalChange;
//    }
//}


void Simulation::applyVelocity(float dt, std::vector<glm::vec2>& velocityChanges)
{
    size_t N = boids.size();
    for (size_t i = 0; i < N; ++i) {
        Boid& b = boids[i];
        b.velocity += velocityChanges[i] * params.slowDownFactor;
        float speed = glm::length(b.velocity);
        if (speed > params.maxSpeed) b.velocity = (b.velocity / speed) * params.maxSpeed;
        b.position += b.velocity * dt;
    }
}

void Simulation::checkEatenPrey()
{
    size_t N = boids.size();
    std::vector<size_t> eatenPreyLocal;

    for (size_t i = 0; i < N; ++i) {
        if (boids[i].type != PREDATOR) continue;
        for (size_t j = 0; j < N; ++j) {
            if (boids[j].type != PREY) continue;
            if (BoidRules::computeEatPrey(i, j, boids, params.predatorEatDistance))
                eatenPreyLocal.push_back(j);
        }
    }

    // Rimuove le prede mangiate
    if (!eatenPreyLocal.empty()) {
        std::sort(eatenPreyLocal.rbegin(), eatenPreyLocal.rend());
        for (size_t idx : eatenPreyLocal) {
            if (idx < boids.size() && boids[idx].type == PREY)
                boids.erase(boids.begin() + idx);
        }
    }
}

void Simulation::spawnNewBoids()
{
    std::vector<std::pair<size_t, size_t>> spawnPairs;
    std::vector<int> boidCouplesLocal;

    size_t N = boids.size();
    for (size_t i = 0; i < N; ++i) {
        if (boids[i].type != PREY) continue;
        BoidRules::computeMating(i, boids, boidCouplesLocal, spawnPairs, params.mateDistance, params.mateThreshold, params.matingAge);
    }

    // Spawn nuovi boid
    for (auto& p : spawnPairs)
        boids.push_back(BoidRules::computeSpawnedBoid(boids[p.first], boids[p.second], currentTime));
}